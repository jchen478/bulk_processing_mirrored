#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "readData.h"
#include "bnei_set.h"
#include "cell.h"
#include "link.h"
#include "contact.h"
#include "ZeroVar.h"
#include "lead.h"
#include "group.h"
#include "errorCheck.h"

using namespace std;

/**
* \file main.cu
* \brief Characterizes aggregates that remain after redispersion
*
* Contain for loop through trajectories and outputs the following
* - Total number of contact (broken + unbroken)
* - Total number of contact, eliminated of joint duplications
*/

int main()
{

	int maxCon = 50;
	int maxGr = 672; 
	int maxBin = 672;
	int npcn = 5000;

	// 1. Read parameters associated with redispersion cycle ...
	//    and other fiber parameters
	int nfib, nseg, config_write, contact_write; 
	int bdimx, bdimy, bdimz;
	float rp, contact_cutoff, rep_cutoff, overlap;
	float dt, strain, sidex, sidey, sidez;
	float fstar, fact, Astar, decatt;
	float dx_ref, dy_ref, dz_ref;

	FILE *aggChar_input;
	aggChar_input= fopen("aggChar_input.txt", "r");

	fscanf(aggChar_input, "%d", &nfib);
	fscanf(aggChar_input, "%*[^\n]%d", &nseg);
	fscanf(aggChar_input, "%*[^\n]%f", &rp);
	fscanf(aggChar_input, "%*[^\n]%f", &contact_cutoff);
	fscanf(aggChar_input, "%*[^\n]%f", &rep_cutoff);
	fscanf(aggChar_input, "%*[^\n]%f", &overlap);
	fscanf(aggChar_input, "%*[^\n]%f", &dt);
	fscanf(aggChar_input, "%*[^\n]%f", &strain);
	fscanf(aggChar_input, "%*[^\n]%f", &sidex);
	fscanf(aggChar_input, " %f", &sidey);
	fscanf(aggChar_input, " %f", &sidez);
	fscanf(aggChar_input, "%*[^\n]%d", &config_write);
	fscanf(aggChar_input, "%*[^\n]%d", &contact_write);
	fscanf(aggChar_input, "%*[^\n]%f", &fstar);
	fscanf(aggChar_input, "%*[^\n]%f", &fact);
	fscanf(aggChar_input, "%*[^\n]%f", &Astar);
	fscanf(aggChar_input, "%*[^\n]%f", &decatt);
	fscanf(aggChar_input, "%*[^\n]%f", &dx_ref);
	fscanf(aggChar_input, "%*[^\n]%f", &dy_ref);
	fscanf(aggChar_input, "%*[^\n]%f", &dz_ref);
	fscanf(aggChar_input, "%*[^\n]%d", &bdimx);
	fscanf(aggChar_input, "%*[^\n]%d", &bdimy);
	fscanf(aggChar_input, "%*[^\n]%d", &bdimz);
	fclose(aggChar_input); 

	// Constant calculations
	int nConfig, nxbinMax, nybinMax, nzbinMax;

	// number of configurations
	nConfig = int(strain / (dt*float(config_write))) + 1;
	
	// maximum bin dimensions
	nxbinMax = int(floorf(sidex / dx_ref));
	nybinMax = int(floorf(sidey / dy_ref));
	nzbinMax = int(floorf(sidez / dz_ref));
	if (nxbinMax % 2 != 0){
		nxbinMax--;
	}
	if (nybinMax % 2 != 0){
		nybinMax--;
	}
	if (nzbinMax % 2 != 0){
		nzbinMax--;
	}

	// cutoffs
	contact_cutoff = powf((contact_cutoff + 2.0), 2.0);
	rep_cutoff = powf((rep_cutoff + 2.0), 2.0);


	// 2. Open trajectory files
	FILE *rxfile, *ryfile, *rzfile; 
	FILE *pxfile, *pyfile, *pzfile;
	rxfile = fopen("rx.txt", "rb");
	ryfile = fopen("ry.txt", "rb");
	rzfile = fopen("rz.txt", "rb");
	pxfile = fopen("px.txt", "rb");
	pyfile = fopen("py.txt", "rb");
	pzfile = fopen("pz.txt", "rb");

	// read simulation case
	FILE *INSInput;
	INSInput = fopen("INSinput.gen","r");
	int simcase;
	fscanf(INSInput, "%d", &simcase);
	fclose(INSInput);	

	// Based on simulation cases, determine simulation
	// box size at every frame
	float *Lx, *Ly, *Lz; 
	Lx = (float*)malloc(nConfig*sizeof(float)); 
	Ly = (float*)malloc(nConfig*sizeof(float));
	Lz = (float*)malloc(nConfig*sizeof(float));
	 
	// case 0 - basis - sidex does not change
	if (simcase == 0){
		for (int f = 0; f < nConfig; f++){
			Lx[f] = sidex; 
			Ly[f] = sidey; 
			Lz[f] = sidez; 
		}
	}
	// case 1 - redispersion - read sidex from Lbox.txt
	else {
		// Read box info
		FILE *BoxFile;
		float dum;
		int nLbox, box_write;
		BoxFile = fopen("box.gen","r");
		fscanf(BoxFile, "%f", &dum);
		fscanf(BoxFile, "%*[^\n]%f", &dum); 
		fscanf(BoxFile, "%*[^\n]%f", &dum); 
		fscanf(BoxFile, "%*[^\n]%f", &dum); 
		fscanf(BoxFile, "%*[^\n]%d", &box_write); 
		fclose(BoxFile); 
		nLbox = strain / (dt *float(box_write)) + 1;

		// Read box dimensions
		FILE *LboxFile;
		float LxTmp, LyTmp, LzTmp;
		LboxFile = fopen("Lbox.txt","r");
		for (int box = 0; box < nLbox; box++){
			fscanf(LboxFile, "%f %f %f %f %f %f %f",
				&dum, &LxTmp, &LyTmp, &LzTmp, &dum, &dum, &dum);
			if ((box*box_write) % config_write == 0){
				Lx[box*box_write/config_write] = LxTmp;
				Ly[box*box_write/config_write] = LyTmp;
				Lz[box*box_write/config_write] = LzTmp;
			}
		}
		fclose(LboxFile); 
	}


	// Open output file
	FILE *ContactFile;
	ContactFile = fopen("ContactStat.txt", "w");

	// 4. Allocate memory for both host and device operations
	int nxbin, nybin, nzbin, num_groups;
	int total_overlap, total_contact, total_contact_no_joints;
	float total_forc, total_dist_inCon;
	float delta_rx, dx, dy, dz;
	float *rx, *ry, *rz, *px, *py, *pz;
	rx = (float *)malloc(nfib*nseg*sizeof(float));
	ry = (float *)malloc(nfib*nseg*sizeof(float));
	rz = (float *)malloc(nfib*nseg*sizeof(float));
	px = (float *)malloc(nfib*nseg*sizeof(float));
	py = (float *)malloc(nfib*nseg*sizeof(float));
	pz = (float *)malloc(nfib*nseg*sizeof(float));

	float *d_rx, *d_ry, *d_rz; 
	float *d_px, *d_py, *d_pz;
	hipMalloc((void**)&d_rx, nfib*nseg*sizeof(float));
	hipMalloc((void**)&d_ry, nfib*nseg*sizeof(float));
	hipMalloc((void**)&d_rz, nfib*nseg*sizeof(float));
	hipMalloc((void**)&d_px, nfib*nseg*sizeof(float));
	hipMalloc((void**)&d_py, nfib*nseg*sizeof(float));
	hipMalloc((void**)&d_pz, nfib*nseg*sizeof(float));

	int *d_nfib, *d_nseg, *d_maxBin;
	int *d_npcn, *d_maxCon, *d_maxGr;
	int *d_total_overlap, *d_total_contact, *d_total_contact_no_joints;
	float *d_total_forc, *d_total_dist_inCon;
	float *d_dx, *d_dy, *d_dz, *d_delta_rx;
	float *d_sidex, *d_sidey, *d_sidez;
	float *d_fstar, *d_fact, *d_Astar, *d_decatt;
	hipMalloc((void**)&d_nfib, sizeof(int));
	hipMalloc((void**)&d_nseg, sizeof(int));
	hipMalloc((void**)&d_maxBin, sizeof(int));
	hipMalloc((void**)&d_maxGr, sizeof(int));
	hipMalloc((void**)&d_npcn, sizeof(int));
	hipMalloc((void**)&d_maxCon, sizeof(int));
	hipMalloc((void**)&d_total_overlap, sizeof(int));
	hipMalloc((void**)&d_total_contact, sizeof(int));
	hipMalloc((void**)&d_total_contact_no_joints, sizeof(int));
	hipMalloc((void**)&d_total_forc, sizeof(float));
	hipMalloc((void**)&d_total_dist_inCon, sizeof(float));
	hipMalloc((void**)&d_dx, sizeof(float));
	hipMalloc((void**)&d_dy, sizeof(float));
	hipMalloc((void**)&d_dz, sizeof(float));
	hipMalloc((void**)&d_sidex, sizeof(float));
	hipMalloc((void**)&d_sidey, sizeof(float));
	hipMalloc((void**)&d_sidez, sizeof(float));
	hipMalloc((void**)&d_delta_rx, sizeof(float));
	hipMalloc((void**)&d_fstar, sizeof(float));
	hipMalloc((void**)&d_fact, sizeof(float));
	hipMalloc((void**)&d_Astar, sizeof(float));
	hipMalloc((void**)&d_decatt, sizeof(float));

	int *bnei, *bin, *list, *bnum;
	int *d_nxbin, *d_nybin, *d_nzbin;
	int *d_bdimx, *d_bdimy, *d_bdimz;
	int *potCon, *potConSize, *d_num_groups;
	int *status, *lead_clist, *nc, *ifiber, *ncnt, *clist_pos, *groupId; 
	hipMalloc((void**)&bnei, bdimx*bdimy*bdimz*nxbinMax*nybinMax*nzbinMax*sizeof(int));
	hipMalloc((void**)&bin, nxbinMax*nybinMax*nzbinMax*sizeof(int));
	hipMalloc((void**)&list, maxBin*nxbinMax*nybinMax*nzbinMax*sizeof(int));
	hipMalloc((void**)&bnum, nfib*nseg*sizeof(int));
	hipMalloc((void**)&d_nxbin, sizeof(int));
	hipMalloc((void**)&d_nybin, sizeof(int));
	hipMalloc((void**)&d_nzbin, sizeof(int));
	hipMalloc((void**)&d_bdimx, sizeof(int));
	hipMalloc((void**)&d_bdimy, sizeof(int));
	hipMalloc((void**)&d_bdimz, sizeof(int));
	hipMalloc((void**)&potCon, nfib*nseg * npcn * sizeof(int));
	hipMalloc((void**)&potConSize, nfib*nseg*sizeof(int));
	hipMalloc((void**)&status, nfib*nseg*sizeof(int));
	hipMalloc((void**)&lead_clist, nfib*nseg*maxGr*sizeof(int));
	hipMalloc((void**)&nc, nfib*nseg*sizeof(int));
	hipMalloc((void**)&clist_pos, nfib*nseg*maxGr*sizeof(int));
	hipMalloc((void**)&ifiber, nfib*nseg * 2 * maxGr*sizeof(int));
	hipMalloc((void**)&ncnt, nfib*nseg*sizeof(int));
	hipMalloc((void**)&d_num_groups, sizeof(int));
	hipMalloc((void**)&groupId, nfib*nseg*sizeof(int));


	int *ncpf, *clist; 
	hipMalloc((void**)&ncpf, (nfib*nseg)*sizeof(int));
	hipMalloc((void**)&clist, nfib*nseg*maxCon*sizeof(int));


	float *d_contact_cutoff, *d_rep_cutoff, *d_over_cut, *d_rp;
	hipMalloc((void**)&d_contact_cutoff, sizeof(float));
	hipMalloc((void**)&d_rep_cutoff, sizeof(float));
	hipMalloc((void**)&d_over_cut, sizeof(float));
	hipMalloc((void**)&d_rp, sizeof(float));

	// copy memory to device
	hipMemcpy(d_bdimx, &bdimx, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_bdimy, &bdimy, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_bdimz, &bdimz, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_maxBin, &maxBin, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_maxCon, &maxCon, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_maxGr, &maxGr, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_npcn, &npcn, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_nseg, &nseg, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_decatt, &decatt, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_fstar, &fstar, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_fact, &fact, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Astar, &Astar, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_contact_cutoff, &contact_cutoff, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_rep_cutoff, &rep_cutoff, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_over_cut, &overlap, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_rp, &rp, sizeof(float), hipMemcpyHostToDevice);


	for (int step = 0; step < nConfig; step++){

		readData(rxfile, rx, nfib*nseg);
		readData(ryfile, ry, nfib*nseg);
		readData(rzfile, rz, nfib*nseg);
		readData(pxfile, px, nfib*nseg);
		readData(pyfile, py, nfib*nseg);
		readData(pzfile, pz, nfib*nseg);

		delta_rx = float(step*config_write) * dt;
		delta_rx -= lroundf(delta_rx / sidex)*sidex;

		sidex = Lx[step]; 
		sidey = Ly[step];
		sidez = Lz[step];

		nxbin = int(floorf(sidex / dx_ref));
		nybin = int(floorf(sidey / dy_ref));
		nzbin = int(floorf(sidez / dz_ref));
		if (nxbin % 2 != 0){
			nxbin--;
		}
		if (nybin % 2 != 0){
			nybin--;
		}
		if (nzbin % 2 != 0){
			nzbin--;
		}
		dx = sidex / float(nxbin);
		dy = sidey / float(nybin);
		dz = sidez / float(nzbin);
		
		total_contact = 0; 
		total_contact_no_joints = 0;
		total_overlap = 0; 
		total_forc = 0.0;
		total_dist_inCon = 0.0; 
		num_groups = 0; 

		// copy memory to device
		hipMemcpy(d_rx, rx, nfib*nseg*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_ry, ry, nfib*nseg*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_rz, rz, nfib*nseg*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_px, px, nfib*nseg*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_py, py, nfib*nseg*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_pz, pz, nfib*nseg*sizeof(float), hipMemcpyHostToDevice);

		hipMemcpy(d_nxbin, &nxbin, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_nybin, &nybin, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_nzbin, &nzbin, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_num_groups, &num_groups, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_total_forc, &total_forc, sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_total_dist_inCon, &total_dist_inCon, sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_total_contact, &total_contact, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_total_contact_no_joints, &total_contact_no_joints, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_total_overlap, &total_overlap, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_dx, &dx, sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_dy, &dy, sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_dz, &dz, sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_sidex, &sidex, sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_sidey, &sidey, sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_sidez, &sidez, sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_delta_rx, &delta_rx, sizeof(float), hipMemcpyHostToDevice);

		// begin characterization
		// 0. zero the variables
		ZeroVar << <nzbin*nybin, nxbin >> >(bin);
		ZeroVar << <nfib, nseg >> >(potConSize);
		ZeroVar << <nfib, nseg >> >(ncpf);
		ZeroVar << <nfib, nseg >> >(status);
		ZeroVar << <nfib, nseg >> >(nc);
		ZeroVar << <nfib, nseg >> >(ncnt);

		// 1. set the neighbors of the bins
		bnei_set << < nzbin*nybin, nxbin >> > (bnei, d_nxbin, d_nybin, d_nzbin,
			d_bdimx, d_bdimy, d_bdimz);

		// 2. Put fibers in the bins
		cell << < nfib/32, nseg*32 >> >(bin, list, bnum, d_rx, d_ry, d_rz, d_nxbin, d_nybin, d_nzbin,
			d_maxBin, d_sidex, d_sidey, d_sidez, d_delta_rx, d_dx, d_dy, d_dz);

		// 3. Find possible contacting pairs and eliminate adjacent segments
		link << < nfib*nseg, bdimx*bdimy*bdimz >> > 
			(bin, list, bnei, bnum, potCon, potConSize, d_npcn, d_nseg, d_nxbin, d_nybin, d_nzbin);

		// 4. check for contacts and obtain contact statistics
		contact << < nfib / 32, nseg * 32 >> >(d_total_contact, d_total_overlap, 
			potCon, potConSize, d_npcn, d_rx, d_ry, d_rz,
			d_px, d_py, d_pz, d_sidex, d_sidey, d_sidez,
			d_delta_rx, d_rp, d_over_cut, d_contact_cutoff, 
			d_rep_cutoff, d_maxCon, ncpf, clist, d_Astar, d_decatt,
			d_fact, d_fstar, d_total_forc, d_total_dist_inCon);

		// 5. Find leaders of each contacting group
		lead << < nfib / 32, nseg * 32 >> >(ncpf, clist, status, nc, lead_clist, d_maxCon, d_maxGr);

		// 6. Eliminate contacts at joints
		group << < nfib / 32, nseg * 32 >>> (ifiber, ncnt, ncpf, d_nseg, 
			clist, status, lead_clist, nc, d_maxCon, clist_pos, d_maxGr,
			groupId, d_num_groups, d_total_contact_no_joints);

		// 5. copy results back to host
		hipMemcpy(&total_contact, d_total_contact, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&total_contact_no_joints, d_total_contact_no_joints, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&total_overlap, d_total_overlap, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&num_groups, d_num_groups, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&total_forc, d_total_forc, sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(&total_dist_inCon, d_total_dist_inCon, sizeof(float), hipMemcpyDeviceToHost);

		// 6. output to file or console
		fprintf(ContactFile, "%10.4f %8d %4d %6.3f %8d %6.3f %4d %10.6f %10.6f\n",
			float(step*config_write)*dt, num_groups, total_contact, float(total_contact) / float(nfib),
			total_contact_no_joints, float(total_contact_no_joints) / float(nfib), total_overlap, 
			total_forc/float(total_contact),total_dist_inCon/float(total_contact));
	}

	fclose(ContactFile);

	fclose(rxfile); fclose(ryfile); fclose(rzfile);
	fclose(pxfile); fclose(pyfile); fclose(pzfile);

	free(rx); free(ry); free(rz); 
	free(px); free(py); free(pz);

	free(Lx); free(Ly); free(Lz); 
	
	hipFree(d_rx); hipFree(d_ry); hipFree(d_rz);
	hipFree(d_px); hipFree(d_py); hipFree(d_pz);

	hipFree(d_nfib); hipFree(d_nseg); hipFree(d_maxBin); hipFree(d_npcn);
	hipFree(d_dx); hipFree(d_dy); hipFree(d_dz); hipFree(d_delta_rx);
	hipFree(d_sidex); hipFree(d_sidey); hipFree(d_sidez); hipFree(d_rp);
	hipFree(d_rep_cutoff); hipFree(d_contact_cutoff); hipFree(d_over_cut);
	hipFree(d_decatt); hipFree(d_fstar); hipFree(d_Astar); hipFree(d_fact);
	hipFree(ncpf); hipFree(clist);  hipFree(d_maxCon);
	hipFree(d_maxGr); 
	hipFree(bnei); hipFree(list); hipFree(bin); hipFree(bnum);
	hipFree(d_nxbin); hipFree(d_nybin); hipFree(d_nzbin);
	hipFree(d_bdimx); hipFree(d_bdimy); hipFree(d_bdimz);
	hipFree(potCon);    hipFree(potConSize);
	hipFree(d_total_contact); hipFree(d_total_overlap); 
	hipFree(d_total_contact_no_joints);
	hipFree(d_total_forc);  
	hipFree(d_total_dist_inCon);

	hipFree(status); hipFree(lead_clist);  hipFree(nc); hipFree(d_num_groups); 
	hipFree(ifiber); hipFree(ncnt); hipFree(clist_pos); hipFree(groupId);

    return 0;
}

