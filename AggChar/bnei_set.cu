#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include "bnei_set.h"

using namespace std;

/**
* \file bnei_set.cu
* \brief Finds neighboring bins for each bin in the simulation
*
* Each bin i has bdimx*bdimy*bdimz neighboring bins. This function 
* finds the index of all neighbors and store in bnei. 
*
* \param bnei list of neighboring bins for each bin
* \param nxbinpt pointer to nxbin, the number of bins in the x-direction
* \param nybinpt pointer to nybin, the number of bins in the y-direction
* \param nzbinpt pointer to nzbin, the number of bins in the z-direction
* \param bdimxpt pointer to bdimx, the number of sub-bins in the x-direction 
* \param bdimypt pointer to bdimy, the number of sub-bins in the y-direction
* \param bdimzpt pointer to bdimz, the number of sub-bins in the z-direction
* 
*/

__global__ void bnei_set(int *bnei, int *nxbinpt, int *nybinpt, int *nzbinpt,
	int *bdimxpt, int *bdimypt, int *bdimzpt){

	int nxbin = *nxbinpt;
	int nybin = *nybinpt;
	int nzbin = *nzbinpt;
	int bdimx = *bdimxpt;
	int bdimy = *bdimypt;
	int bdimz = *bdimzpt;

	int tid = threadIdx.x + blockIdx.x*blockDim.x;

	int xbin, ybin, zbin, xcen, ycen, zcen;
	
	int xpos, ypos, zpos;
	int xind, yind, zind;
	int xdiff, ydiff, zdiff;
	int ind;

	int tid2; 

	xcen = (bdimx - 1) / 2; 
	ycen = (bdimy - 1) / 2;
	zcen = (bdimz - 1) / 2;

	zbin = tid / (nxbin*nybin);
	ybin = (tid - zbin*nxbin*nybin) / nxbin;
	xbin = tid - ybin*nxbin - zbin*nxbin*nybin;

	for (tid2 = 0; tid2 < bdimx*bdimy*bdimz; tid2++){

		zpos = tid2 / (bdimx*bdimy);
		ypos = (tid2 - zpos*bdimx*bdimy) / bdimx;
		xpos = tid2 - ypos*bdimx - zpos*bdimx*bdimy;

		xdiff = xpos - xcen;
		xind = xbin + xdiff;
		if (xind < 0){
			xind += nxbin;
		}
		if (xind >= nxbin){
			xind -= nxbin;
		}
		ydiff = ypos - ycen;
		yind = ybin + ydiff;
		if (yind < 0){
			yind += nybin;
		}
		if (yind >= nybin){
			yind -= nybin;
		}
		zdiff = zpos - zcen;
		zind = zbin + zdiff;
		if (zind < 0){
			zind += nzbin;
		}
		if (zind >= nzbin){
			zind -= nzbin;
		}

		ind = xind + yind*nxbin + zind*nxbin*nybin;

		if ((xbin + ybin*nxbin + zbin*nxbin*nybin + tid2 * nxbin*nybin*nzbin) >= 0 && (xbin + ybin*nxbin + zbin*nxbin*nybin + tid2 * nxbin*nybin*nzbin) < nxbin*nybin*nzbin*bdimx*bdimy*bdimz){			
			bnei[xbin + ybin*nxbin + zbin*nxbin*nybin + tid2 * nxbin*nybin*nzbin] = ind;
		}
		else{
			printf("tid2 %4d\n", tid2);
		}		
	}	
}
